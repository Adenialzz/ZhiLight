#include "hip/hip_runtime.h"
// Author: Gaojunmin@zhihu.com

#include "nn/quant/gptq/gptq.h"
#include <bmengine/core/core.h>
#include <bmengine/functions/all.h>
#include <bmengine/logger/std_log_op.hpp>
#include <assert.h>

#include <cstdint>
#include <cstdio>

#include <hip/hip_runtime.h>

#define MMA_M 16
#define MMA_N 8
#define MMA_K 32

#define CHUNK_K 2  // BLOCK_K / MMA_K

#define WARP_SIZE 32

#define THREAD_COPY_BYTES 16

// 一行64字节
#define CHUNK_LINE_BYTES 64          // CHUNK_K * MMA_K * sizeof(half)
// 一个 warp 一次可以拷8行。
#define CHUNK_COPY_LINES_PER_WARP 8  // WARP_SIZE * THREAD_COPY_BYTES / CHUNK_LINE_BYTES
// 一行需要 4个 lane
#define CHUNK_COPY_LINE_LANES 4      // WARP_SIZE / CHUNK_COPY_LINES_PER_WARP

#define AB_SMEM_STRIDE 64  // CHUNK_K * MMA_K

#define C_SMEM_STRIDE 136  // BLOCK_N + 8 of int

#define BLOCK_STRIDE 16

// 两行共 128 字节
#define SMEM_BANK_ROWS 2  // 32 * 4 / (AB_SMEM_STRIDE * sizeof(half))

#define PERMUTED_OFFSET 16  // bytes
#define PERMUTED_COLS 8

#define LDMATRIX_X2(R0, R1, addr) \
    asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))

#define LDMATRIX_X4(R0, R1, R2, R3, addr)                                             \
    asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" \
                 : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3)                             \
                 : "r"(addr))

#define MMA16832(D0, D1, D2, D3, A0, A1, A2, A3, B0, B1)                                 \
    asm volatile(                                                                        \
        "mma.sync.aligned.m16n8k32.row.col.s32.s8.u8.s32 {%0,%1,%2,%3}, {%4,%5,%6,%7}, " \
        "{%8,%9}, {%10,%11,%12,%13};\n"                                                  \
        : "=r"(D0), "=r"(D1), "=r"(D2), "=r"(D3)                                         \
        : "r"(A0), "r"(A1), "r"(A2), "r"(A3), "r"(B0), "r"(B1),                          \
          "r"(D0), "r"(D1), "r"(D2), "r"(D3))

#define CP_ASYNC_CG(dst, src, Bytes) \
    asm volatile("cp.async.cg.shared.global [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(Bytes))

#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)

#define CP_ASYNC_WAIT_GROUP(N) asm volatile("cp.async.wait_group %0;\n" ::"n"(N))

using namespace bmengine;

typedef int32_t OUT_T;
template<
    uint32_t BLOCK_M = 256,
    uint32_t BLOCK_N = 128,
    uint32_t BLOCK_K = 64,
    uint32_t NUM_WARPS_M = 4,
    uint32_t NUM_WARPS_N = 2,
    int K_STAGE = 3,
    int GROUP_SIZE=128
>
__global__ void KERNEL_gemm_groupwise_int8(
    const int8_t *__restrict__ A,
    const int8_t *__restrict__ B,
    half *__restrict__ C,
    int8_t *__restrict__ q_scales,     // (N, K / group_size)
    half *__restrict__ scales_scales,  // (N)
    float *__restrict__ A_scales,      // (M)
    uint32_t M, uint32_t N, uint32_t K
) {
#if __CUDA_ARCH__ >= 800
    static constexpr uint32_t WARP_M = BLOCK_M / NUM_WARPS_M;
    static constexpr uint32_t WARP_N = BLOCK_N / NUM_WARPS_N;
    static constexpr uint32_t NUM_WARPS = NUM_WARPS_M * NUM_WARPS_N;
    static constexpr uint32_t WARP_TILES_M = WARP_M / MMA_M;
    static constexpr uint32_t WARP_TILES_N = WARP_N / MMA_N;

    int RC[WARP_TILES_M][WARP_TILES_N][4];
    uint32_t RA[WARP_TILES_M][4];
    uint32_t RB[WARP_TILES_N][2];

    const uint32_t block_tile_i = (blockIdx.z % 2) ? gridDim.y - blockIdx.y - 1 : blockIdx.y;
    const uint32_t block_tile_j = (blockIdx.z * gridDim.x + blockIdx.x);

    if (block_tile_i * BLOCK_M >= M || block_tile_j * BLOCK_N >= N) {
        return;
    }

    extern __shared__ __align__(16) int8_t smem[][AB_SMEM_STRIDE];

    const uint32_t warp_id = threadIdx.x / WARP_SIZE;
    const uint32_t lane_id = threadIdx.x % WARP_SIZE;

    static constexpr uint32_t B_smem_idx_off = BLOCK_M;
    static constexpr uint32_t smem_stage_off = BLOCK_M + BLOCK_N;

#pragma unroll
    for (int i = 0; i < WARP_TILES_M; ++i) {
#pragma unroll
        for (int j = 0; j < WARP_TILES_N; ++j) {
            RC[i][j][0] = 0;
            RC[i][j][1] = 0;
            RC[i][j][2] = 0;
            RC[i][j][3] = 0;
        }
    }

    const int8_t *A_warp_ptr = &A[block_tile_i * BLOCK_M * K] + BLOCK_M / NUM_WARPS * K * warp_id;
    const int8_t *B_warp_ptr = &B[block_tile_j * BLOCK_N * K] + BLOCK_N / NUM_WARPS * K * warp_id;

    static constexpr int A_smem_iters = BLOCK_M / (CHUNK_COPY_LINES_PER_WARP * NUM_WARPS);
    static constexpr int B_smem_iters = BLOCK_N / (CHUNK_COPY_LINES_PER_WARP * NUM_WARPS);

    uint32_t smem_store_idx = 0;
    uint32_t smem_load_idx = 0;

    uint32_t smem_store_off = 0;
    uint32_t smem_load_off = 0;

    uint32_t shared_addr0 = __cvta_generic_to_shared(&smem[0][0]);

    const uint32_t row = lane_id / CHUNK_COPY_LINE_LANES;
    const uint32_t col = lane_id % CHUNK_COPY_LINE_LANES;
    const uint32_t col_perm = (col + row / SMEM_BANK_ROWS) % CHUNK_COPY_LINE_LANES;
    auto fn_copy_A = [&](const int8_t* A_ptr) {

        int4* A_lane_ptr = (int4 *)(A_ptr + row * K) + col;

        uint32_t A_smem_idx = smem_store_off + BLOCK_M / NUM_WARPS * warp_id;  // warp 从哪一行开始
        A_smem_idx += row; // lane 从哪一行开始

        uint32_t shared_addr = shared_addr0 + A_smem_idx * CHUNK_LINE_BYTES +
                               col_perm * THREAD_COPY_BYTES;

#pragma unroll
        for (int i = 0; i < A_smem_iters; ++i) {
            CP_ASYNC_CG(shared_addr, A_lane_ptr, THREAD_COPY_BYTES);

            A_lane_ptr = (int4 *)((int8_t *)A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            shared_addr += CHUNK_COPY_LINES_PER_WARP * AB_SMEM_STRIDE;
        }
    };

    auto fn_copy_B = [&](const int8_t* B_ptr) {
        int4* B_lane_ptr = (int4 *)(B_ptr + row * K) + col;
        uint32_t B_smem_idx = smem_store_off + B_smem_idx_off + BLOCK_N / NUM_WARPS * warp_id;
        B_smem_idx += row;

        uint32_t shared_addr = shared_addr0 + B_smem_idx * CHUNK_LINE_BYTES +
                               col_perm * THREAD_COPY_BYTES;
#pragma unroll
        for (int i = 0; i < B_smem_iters; ++i) {

            CP_ASYNC_CG(shared_addr, B_lane_ptr, THREAD_COPY_BYTES);

            B_lane_ptr = (int4 *)((int8_t *)B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            shared_addr += CHUNK_COPY_LINES_PER_WARP * AB_SMEM_STRIDE;
        }
    };

    auto fn_copy_AB = [&](uint32_t k_offset) {
        smem_store_idx = (smem_store_idx + 1) % K_STAGE;
        smem_store_off = smem_store_idx * smem_stage_off;

        fn_copy_A(A_warp_ptr + k_offset);
        fn_copy_B(B_warp_ptr + k_offset);

        CP_ASYNC_COMMIT_GROUP();
    };

    // Copy 1
    fn_copy_A(A_warp_ptr);
    fn_copy_B(B_warp_ptr);

    CP_ASYNC_COMMIT_GROUP();

    // Copy 2
    fn_copy_AB(CHUNK_K * MMA_K);

    // Copy 3
    if (K_STAGE > 3) {
        fn_copy_AB(2 * CHUNK_K * MMA_K);
    }
    if (K_STAGE > 4) {
        fn_copy_AB(3 * CHUNK_K * MMA_K);
    }
    if (K_STAGE > 5) {
        fn_copy_AB(4 * CHUNK_K * MMA_K);
    }

    const uint32_t col_perm_offset_A = (lane_id / 16) * 16 + (lane_id % 16 /*% (PERMUTED_COLS * SMEM_BANK_ROWS)*/)
                                                             / SMEM_BANK_ROWS * PERMUTED_OFFSET;
    auto fn_load_A = [&](uint32_t offset=0) {
        // ldmatrix 一次加载 m8n8 个 b16 ；即一次加载 8 X 16Bytes；需要八个地址。
        // 即 LOAD_K = 8; MMA_K = 16; 所以矩阵要拆成两半/4个。左上下，右上下。
        // --每四个线程(lane_id % 8) 加载 4 * 4 = 16 Bytes; 即对应 m8n8 中的一行。
        // 地址：左半个矩阵16行地址；右半16行地址; 共 32
        // permute 之前是(lane_id / 16) * 16
        uint32_t lane_dim2 = (offset + col_perm_offset_A) % AB_SMEM_STRIDE;
        uint32_t A_smem_idx = smem_load_off + (warp_id / NUM_WARPS_N) * WARP_M;
        uint32_t A_smem_lane_addr = shared_addr0 + (A_smem_idx + lane_id % 16) * CHUNK_LINE_BYTES + lane_dim2;
#pragma unroll
        for (uint32_t i = 0; i < WARP_TILES_M; ++i) {
            LDMATRIX_X4(RA[i][0], RA[i][1], RA[i][2], RA[i][3],
                        A_smem_lane_addr + i * MMA_M * AB_SMEM_STRIDE);
        }
    };

    const uint32_t col_perm_offset_B = ((lane_id / 8) % 2) * 16 +
                                       (lane_id /*% 8*/ % (PERMUTED_COLS * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS * PERMUTED_OFFSET;

    auto fn_load_B = [&](uint32_t offset=0) {
        uint32_t lane_dim2 = (offset + col_perm_offset_B) % AB_SMEM_STRIDE;
        uint32_t B_smem_idx = smem_load_off + B_smem_idx_off + (warp_id % NUM_WARPS_N) * WARP_N;
        uint32_t B_smem_lane_addr = shared_addr0 + (B_smem_idx + lane_id % 8) * CHUNK_LINE_BYTES + lane_dim2;
#pragma unroll
        for (uint32_t j = 0; j < WARP_TILES_N; ++j) {
            LDMATRIX_X2(RB[j][0], RB[j][1], B_smem_lane_addr + j * MMA_N * AB_SMEM_STRIDE);
        }
    };

    auto fn_load_AB = [&](int offset=0) {
        fn_load_A(offset);
        fn_load_B(offset);
    };

    auto fn_mma = [&]() {
#pragma unroll
        for (int i = 0; i < WARP_TILES_M; ++i) {
#pragma unroll
            for (int j = 0; j < WARP_TILES_N; ++j) {
                int j_s = (i % 2) ? (WARP_TILES_N - j - 1) : j;

                MMA16832(RC[i][j_s][0], RC[i][j_s][1], RC[i][j_s][2], RC[i][j_s][3],
                          RA[i][0], RA[i][1], RA[i][2], RA[i][3], 
                          RB[j_s][0], RB[j_s][1]);
            }
        }
    };

    int compute_count = 0;
    // warp_col + lane_col, 2 values per lane
    uint32_t col_base = (warp_id % NUM_WARPS_N) * WARP_N + (lane_id % 4) * 2;

    int cur_scale[2];
    int next_scale[2];
    int K_GRP = K / GROUP_SIZE;
    auto fn_rescale = [&](bool last=false) {
        int g_idx = compute_count / (GROUP_SIZE / BLOCK_K);
#pragma unroll
        for (int j = 0; j < WARP_TILES_N; ++j) {
            auto n0 = col_base + j * MMA_N;
            cur_scale[0] = q_scales[n0 * K_GRP + g_idx - 1];
            cur_scale[1] = q_scales[(n0 + 1) * K_GRP + g_idx - 1];
            if (!last) {
                next_scale[0] = q_scales[n0 * K_GRP + g_idx];
                next_scale[1] = q_scales[(n0 + 1) * K_GRP + g_idx];
            }
#pragma unroll
            for (int i = 0; i < WARP_TILES_M; ++i) {
                if (last) {
                    RC[i][j][0] = RC[i][j][0] * cur_scale[0];
                    RC[i][j][1] = RC[i][j][1] * cur_scale[1];
                } else {
                    RC[i][j][0] = RC[i][j][0] * cur_scale[0] / next_scale[0];
                    RC[i][j][1] = RC[i][j][1] * cur_scale[1] / next_scale[1];
                }
            }
        }
    };

    CP_ASYNC_WAIT_GROUP(K_STAGE - 2);
    __syncthreads();
    fn_load_AB(0);

    auto fn_compute_before = [&]() {
        fn_mma();
        fn_load_AB(MMA_K);
    };
    auto fn_compute_after = [&]() {
        fn_mma();
        fn_load_AB(0);
        compute_count++;
        if (compute_count % (GROUP_SIZE / BLOCK_K) == 0) {
            fn_rescale();
        }
    };
    /* --------------------------------------------------------------------- */
    /* -------------------------- MAIN-LOOP over K ------------------------- */
    /* --------------------------------------------------------------------- */
    const uint32_t K_tiles = K / MMA_K;
    for (uint32_t tile_k = CHUNK_K * (K_STAGE - 1); tile_k < K_tiles; tile_k += CHUNK_K) {
        fn_compute_before();

        fn_copy_AB(tile_k * MMA_K);

        smem_load_idx = (smem_load_idx + 1) % K_STAGE;
        smem_load_off = smem_load_idx * smem_stage_off;

        CP_ASYNC_WAIT_GROUP(K_STAGE - 2);
        __syncthreads();

        fn_compute_after();
    } // End of MAIN-LOOP

    // same as MAIN-LOOP without copy
#pragma unroll
    for (int stage = K_STAGE - 3; stage >= 0; stage--) {
        fn_compute_before();

        smem_load_idx = (smem_load_idx + 1) % K_STAGE;
        smem_load_off = smem_load_idx * smem_stage_off;

        if (stage == 0) {
            CP_ASYNC_WAIT_GROUP(0);
        } else if (stage == 1) {
            CP_ASYNC_WAIT_GROUP(1);
        } else if (stage == 2) {
            CP_ASYNC_WAIT_GROUP(2);
        } else {
            CP_ASYNC_WAIT_GROUP(3);
        }
        __syncthreads();

        fn_compute_after();
    } // End

#pragma unroll
    for (int k_step = 1; k_step < CHUNK_K; ++k_step) {
        fn_compute_before();
    }

    fn_mma();
    fn_rescale(true);

    __syncthreads();

    // Write output
    uint32_t warp_lane_row = (warp_id / NUM_WARPS_N) * WARP_M
                             + (lane_id / 4);
    half *smem_warp_tile_row_ptr = (half*)(&smem[0][0]) + warp_lane_row * C_SMEM_STRIDE;

    uint32_t col1 = (warp_id % NUM_WARPS_N) * WARP_N +
                    (lane_id % 4) * (sizeof(uint32_t) / sizeof(half));
#pragma unroll
    for (uint32_t i = 0; i < WARP_TILES_M; ++i) {
        float A_scale0 = A_scales[warp_lane_row + i * MMA_M];
        float A_scale8 = A_scales[warp_lane_row + i * MMA_M + 8];
#pragma unroll
        for (uint32_t j = 0; j < WARP_TILES_N; ++j) {
            // 上 8 行，4个lane 一行
            half *lane_ptr0 = smem_warp_tile_row_ptr + (i * MMA_M * C_SMEM_STRIDE + col1 + j * MMA_N);
            // 下 8 行
            half *lane_ptr8 = lane_ptr0 + 8 * C_SMEM_STRIDE;

            half B_scale[2];
            half h_ret[2];
            *(int*)(&B_scale[0]) = *(int*)(scales_scales + col1  + j * MMA_N);
            h_ret[0] = float(RC[i][j][0]) * A_scale0 * float(B_scale[0]);
            h_ret[1] = float(RC[i][j][1]) * A_scale0 * float(B_scale[1]);
            *((int *)(lane_ptr0)) = *(int*)(&h_ret[0]);
            half h_ret8[2];
            *(int*)(&B_scale[0]) = *(int*)(scales_scales + col1  + j * MMA_N);
            h_ret8[0] = float(RC[i][j][2]) * A_scale8 * float(B_scale[0]);
            h_ret8[1] = float(RC[i][j][3]) * A_scale8 * float(B_scale[1]);
            *((int *)(lane_ptr8)) = *(int*)(&h_ret8[0]);
        }
    }

    __syncthreads();

    static_assert(SMEM_BANK_ROWS == 2); // 下面的2都是这个
    static constexpr uint32_t WRITE_ITER = (BLOCK_M / NUM_WARPS / 2);
    const uint32_t gmem_idx = (block_tile_i * BLOCK_M + warp_id * WRITE_ITER * 2) * N + block_tile_j * BLOCK_N;
    const half *smem_warp_stream_ptr = (half*)(&smem[0][0]) + (warp_id * WRITE_ITER * 2 * C_SMEM_STRIDE
                                       + (lane_id / 16) * C_SMEM_STRIDE + (lane_id % 16) * 8);
    const half *lane_C = &C[gmem_idx] + ((lane_id / 16) * N + (lane_id % 16) * 8);

    for (uint32_t i = 0; i < WRITE_ITER; ++i) {
        *((int4 *)(lane_C + i * 2 * N)) = *(int4 *)(smem_warp_stream_ptr + i * 2 * C_SMEM_STRIDE);
    }
#endif
}

#define BLOCK_M 256
#define BLOCK_N 128
#define BLOCK_K 64

#define BLOCK_WARPS_M 4  // BLOCK_M / WARP_M
#define BLOCK_WARPS_N 2  // BLOCK_N / WARP_N

#define K_STAGE 3

static size_t initMmaNew() {
    int dev_id = 0;
    BM_CUDART_ASSERT(hipGetDevice(&dev_id));

    hipDeviceProp_t dev_prop;
    BM_CUDART_ASSERT(hipGetDeviceProperties(&dev_prop, dev_id));

    size_t ab_smem_size = (BLOCK_M + BLOCK_N) * AB_SMEM_STRIDE * sizeof(int8_t) * K_STAGE;
    size_t out_smem_size = BLOCK_M * C_SMEM_STRIDE * sizeof(half);
    size_t smem_max_size = std::max(ab_smem_size, out_smem_size);

    BM_ASSERT_LE(smem_max_size, dev_prop.sharedMemPerMultiprocessor, "");
    auto kernel = KERNEL_gemm_groupwise_int8<BLOCK_M, BLOCK_N, BLOCK_K, BLOCK_WARPS_M, BLOCK_WARPS_N, K_STAGE>;
    BM_CUDART_ASSERT(
        hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_max_size));

    return smem_max_size;
}

inline __device__ __host__ size_t div_ceil(size_t a, size_t b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

void gemm_w4_int8(hipStream_t stream, int8_t *A, int8_t *B, half *C, size_t M, size_t N, size_t K) {
    static size_t smem_max_size = initMmaNew();

    dim3 block(BLOCK_WARPS_M * BLOCK_WARPS_N * WARP_SIZE);
    dim3 grid(BLOCK_STRIDE, div_ceil(M, BLOCK_M), div_ceil(N, BLOCK_N * BLOCK_STRIDE));

    auto kernel = KERNEL_gemm_groupwise_int8<BLOCK_M, BLOCK_N, BLOCK_K, BLOCK_WARPS_M, BLOCK_WARPS_N, K_STAGE>;
    kernel<<<grid, block, smem_max_size, stream>>>(A, B, C,
                                           0, 0, 0,
                                           M, N, K);
    BM_CUDART_ASSERT(hipGetLastError());
}

namespace nn::gptq {
core::Tensor gemm_groupwise_int8(
    const core::Context& ctx,
    const core::Tensor& a,        // (M, K) int8
    const core::Tensor& b,        // (N, K) int8
    const core::Tensor& q_scales, // (N, K / group_size) int8
    const core::Tensor& s_scales, // (N) half
    const core::Tensor& a_scales, // (M) float
    const core::Tensor *bias
) {
    BM_ASSERT((a.mem_bytes() / a.size(-1) % BLOCK_M) == 0, "Wrong input size 0");
    const size_t M = a.mem_bytes() / a.size(-1);
    const size_t K = a.size(1);
    const size_t N = b.size(-2);
    const size_t GROUP_SIZE = K / q_scales.size(-1);

    BM_ASSERT_EQ(2, a.ndim(), "Wrong input dim");
    // BM_ASSERT((M % BLOCK_M) == 0, "Wrong input size 0"); // TODO: support other size
    BM_ASSERT((N % BLOCK_N) == 0, "Wrong weight size 0");

    BM_ASSERT_EQ(a.dtype(), core::DataType::kInt8, "A must be int8");
    BM_ASSERT_EQ(b.dtype(), core::DataType::kInt8, "A must be int8");
    BM_ASSERT_EQ(q_scales.dtype(), core::DataType::kInt8, "q_scales must be int8");
    BM_ASSERT_EQ(s_scales.dtype(), core::DataType::kHalf, "s_scales must be half");
    BM_ASSERT_EQ(a_scales.dtype(), core::DataType::kFloat, "A must be float");

    auto stream = ctx.current_stream()->ptr;
    auto c = ctx.tensor({a.size(0), N}, core::DataType::kHalf);

    dim3 block(BLOCK_WARPS_M * BLOCK_WARPS_N * WARP_SIZE);
    dim3 grid(BLOCK_STRIDE, div_ceil(M, BLOCK_M), div_ceil(N, BLOCK_N * BLOCK_STRIDE));

    size_t ab_smem_size = (BLOCK_M + BLOCK_N) * AB_SMEM_STRIDE * sizeof(int8_t) * K_STAGE;
    size_t out_smem_size = BLOCK_M * C_SMEM_STRIDE * sizeof(half);
    size_t smem_max_size = std::max(ab_smem_size, out_smem_size);
    BM_ASSERT_LE(smem_max_size, ctx.get_max_shared_memory(), "");

    auto kernel = KERNEL_gemm_groupwise_int8<BLOCK_M, BLOCK_N, BLOCK_K, BLOCK_WARPS_M, BLOCK_WARPS_N, K_STAGE>;
    BM_CUDART_ASSERT(hipFuncSetAttribute(reinterpret_cast<const void*>(
        kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_max_size));
    kernel<<<grid, block, smem_max_size, stream>>>(
        a.data<int8_t>(),
        b.data<int8_t>(),
        c.data<half>(),
        q_scales.data<int8_t>(), // TODO: int16
        s_scales.data<half>(),
        a_scales.data<float>(),
        M, N, K);
    BM_CUDART_ASSERT(hipGetLastError());

    return c;
}
} // namespace nn::gptq
